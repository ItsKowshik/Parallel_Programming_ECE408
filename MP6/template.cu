#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//The global function for the final addition of the scan
__global__ void totalAdd(float *input, float *output, float *sum, int len){
  //Starting ixes
  int t = threadIdx.x;
  int start = 2*blockIdx.x*blockDim.x + t;

  //Shared memory for the final add
  __shared__ float addsum;
  if (t == 0)
  {
    if (blockIdx.x == 0){
      addsum = 0;
    }
    else{
      addsum = sum[blockIdx.x - 1]; 
    }
  }
  //Sync threads befor writing to global memory
  __syncthreads();
  if (start < len){
    output[start] = input[start] + addsum;
  }
  if ((start + blockDim.x) < len){
    output[start + blockDim.x] = input[start + blockDim.x]  + addsum;
  }
}

__global__ void scan(float *input, float *output, int len, int boolScan) {
  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from the host

  __shared__ float partialSum[2*BLOCK_SIZE];

  /* Load elements from global */
  int t = threadIdx.x;
  int start;
  int startStride;

  // Determine the starting point and stride based on boolScan
  if (boolScan == 0){
    start = 2*blockIdx.x*blockDim.x + t;
    startStride = blockDim.x;
  }
  else{
    start = 2*blockDim.x*(t+1) - 1;
    startStride = 2*blockDim.x;
  }

  // Load elements into shared memory, handling boundary conditions
  if(start < len){
    partialSum[t] = input[start];
  }
  else{
    partialSum[t] = 0;
  }
  if((start + startStride) < len){
    partialSum[t+blockDim.x] = input[start + startStride];
  }
  else{
    partialSum[t+blockDim.x] = 0;
  }

  /* first scan */
  int stride = 1;
  while (stride < 2*BLOCK_SIZE)
  {
    __syncthreads();
    int ix = (t+1)*stride*2 - 1;
    if ((ix < 2*BLOCK_SIZE) && ((ix-stride) >= 0)){
      partialSum[ix] += partialSum[ix-stride];
    }
    stride *= 2;
  }

  /* second scan */
  stride = BLOCK_SIZE/2;
  while (stride > 0)
  {
    __syncthreads();
    int ix = (t+1)*stride*2 - 1;
    if ((ix + stride) < 2*BLOCK_SIZE){
      partialSum[ix + stride] += partialSum[ix];
    }
    stride /= 2;
  }
  
  /* store back to global */
  __syncthreads();
  int storeStart = 2*blockIdx.x*blockDim.x + t;
  if (storeStart < len){
    output[storeStart] = partialSum[t];
  }
  if ((storeStart + blockDim.x) < len){
    output[storeStart + blockDim.x] = partialSum[t + blockDim.x];
  }

}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  float *deviceScanBlock;
  float *deviceScanSum;
  int numElements; // number of elements in the list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);

  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceScanBlock, numElements * sizeof(float)));  // Store the partial sums of each independent block
  wbCheck(hipMalloc((void **)&deviceScanSum, 2 * BLOCK_SIZE * sizeof(float))); // Store the add-up sums of blocks
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Clearing output memory.");
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbTime_stop(GPU, "Clearing output memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions 
  dim3 dimGrid(ceil(numElements / (2.0*BLOCK_SIZE)), 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  scan<<<dimGrid, dimBlock>>>(deviceInput, deviceScanBlock, numElements, 0);
  hipDeviceSynchronize();

  scan<<<1, dimBlock>>>(deviceScanBlock, deviceScanSum, numElements, 1);
  hipDeviceSynchronize();

  totalAdd<<<dimGrid, dimBlock>>>(deviceScanBlock, deviceOutput, deviceScanSum, numElements);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                     hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  hipFree(deviceScanBlock);
  hipFree(deviceScanSum);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}