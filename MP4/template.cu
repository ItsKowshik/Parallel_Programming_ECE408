#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define TILE_WIDTH 4
#define MASK_WIDTH 3

//@@ Define constant memory for device kernel here
__constant__ float M[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];
__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  __shared__ float input_ds[TILE_WIDTH+2][TILE_WIDTH+2][TILE_WIDTH+2];

 int tx = threadIdx.x;
 int ty = threadIdx.y;
 int tz = threadIdx.z;
 int bx = blockIdx.y * TILE_WIDTH + ty;
 int by = blockIdx.x * TILE_WIDTH + tx;
 int bz = blockIdx.z * TILE_WIDTH + tz;

 int bx_i = bx - 1;       
 int by_i = by - 1;
 int bz_i = bz - 1;

 float sum = 0.0f;

 // Set the halo elements to 0.
 if((bx_i >= 0) && (bx_i < y_size) && (by_i >= 0) && (by_i < x_size) && (bz_i >= 0) && (bz_i < z_size)){
     input_ds[tz][ty][tx] = input[bz_i * x_size * y_size + bx_i * x_size + by_i];
   }
 else{
     input_ds[tz][ty][tx] = 0.0f;
   }
  __syncthreads();

  if((tx < TILE_WIDTH) && (ty < TILE_WIDTH) && (tz < TILE_WIDTH)){
    for(int i = 0; i < MASK_WIDTH; i++){
      for(int j = 0; j < MASK_WIDTH; j++){
        for(int k = 0; k < MASK_WIDTH; k++){
          sum += M[i][j][k] * input_ds[i+tz][j+ty][k+tx];
        }
      }
    }
    if((bx < y_size) && (by < x_size) && (bz < z_size)){
    output[bz * x_size * y_size + bx * x_size + by] = sum;
    }
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =(float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  int input_size = (inputLength - 3)*sizeof(float);
  hipMalloc((void **)&deviceInput, input_size);
  hipMalloc((void **)&deviceOutput, input_size);
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  hipMemcpy(deviceInput, &hostInput[3], input_size, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(M), hostKernel, kernelLength * sizeof(float));
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 dimBlock(TILE_WIDTH + MASK_WIDTH - 1, TILE_WIDTH + MASK_WIDTH - 1, TILE_WIDTH + MASK_WIDTH - 1);
  dim3 dimGrid(ceil(x_size / (1.0 * TILE_WIDTH)),ceil(y_size / (1.0 * TILE_WIDTH)), ceil(z_size / (1.0 * TILE_WIDTH)));
  conv3d<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  //@@ Launch the GPU kernel here
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(&hostOutput[3], deviceOutput, input_size, hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
